/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
int main(){
	int len=16;
	uint32 *hx,*dx;
	uint64 *ht,*dt;

	hipHostMalloc(&hx,len*sizeof(uint32));
	hipHostMalloc(&ht,len*sizeof(uint64));

	hipMalloc(&dx,len*sizeof(uint32));
	hipMalloc(&dt,len*sizeof(uint64));

  for(int i=0;i<len;i++){
		hx[i]=111;		//32 bits random number
	 	ht[i]=0;
		//cout<<"test"<<hx[i];   //for test the function "rand"
	}
  	hipMemcpy(dx,hx,len*sizeof(uint32),hipMemcpyHostToDevice);
	ntt_16_1(dt,dx);
	hipMemcpy(ht,dt,len*sizeof(uint64),hipMemcpyDeviceToHost);

	for(int i=0;i<len;i++){
		cout<<ht[i]<<endl;
	}
	hipFree(dx);
	hipFree(dt);
	hipHostFree(hx);
	hipHostFree(ht);
	return 0;
}
