/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
#define len 4096

int main(){
	const ZZ P=to_ZZ(0xffffffff00000001);
	const ZZ root_4K=to_ZZ((uint64)10974926054405199669);

	uint32 *hx,*dx;
	uint64 *ht,*dt;
	uint64 *h_roots,*d_roots;

	hipHostMalloc(&hx,len*sizeof(uint32));
	hipHostMalloc(&ht,len*sizeof(uint64));
	hipHostMalloc(&h_roots,len*sizeof(uint64));

	hipMalloc(&dx,len*sizeof(uint32));
	hipMalloc(&dt,len*sizeof(uint64));
	hipMalloc(&d_roots,len*sizeof(uint64));

  for(int i=0;i<len;i++){
		hx[i]=i;		//32 bits random number
	 	ht[i]=0;
	}
  for(int i=0;i<64;i++){
		for(int k=0;k<64;k++){
				conv(h_roots[64*i+k],PowerMod(root_4K,i*k,P));
		}
	}
  	hipMemcpy(dx,hx,len*sizeof(uint32),hipMemcpyHostToDevice);
	hipMemcpy(d_roots,h_roots,len*sizeof(uint64),hipMemcpyHostToDevice);
  	ntt_4K_1(dt,dx,d_roots);
	hipMemcpy(ht,dt,len*sizeof(uint64),hipMemcpyDeviceToHost);

	for(int i=0;i<len;i++){
		cout<<ht[i]<<endl;
	}
	hipFree(dx);
	hipFree(dt);
	hipFree(d_roots);
	hipHostFree(hx);
	hipHostFree(ht);
	hipHostFree(h_roots);
	return 0;
}
