/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
#define len 256

int main(){
	const ZZ P=to_ZZ(0xffffffff00000001);
	const ZZ root_256=to_ZZ((uint64)14041890976876060974);

	uint32 *hx,*dx;
	uint64 *ht,*dt;
	uint64 *h_roots,*d_roots;

	hipHostMalloc(&hx,len*sizeof(uint32));
	hipHostMalloc(&ht,len*sizeof(uint64));
	hipHostMalloc(&h_roots,len*sizeof(uint64));

	hipMalloc(&dx,len*sizeof(uint32));
	hipMalloc(&dt,len*sizeof(uint64));
	hipMalloc(&d_roots,len*sizeof(uint64));

  for(int i=0;i<len;i++){
		hx[i]=i;	
	 	ht[i]=0;
	}
  for(int i=0;i<16;i++){
		for(int k=0;k<16;k++){
				conv(h_roots[16*i+k],PowerMod(root_256,i*k,P));
		//test:		cout<<h_roots[16*i+k]<<endl;
		}
	}
  	hipMemcpy(dx,hx,len*sizeof(uint32),hipMemcpyHostToDevice);
  	hipMemcpy(d_roots,h_roots,len*sizeof(uint64),hipMemcpyHostToDevice);
  	ntt_256(dt,dx,d_roots);
  	hipMemcpy(ht,dt,len*sizeof(uint64),hipMemcpyDeviceToHost);

	for(int i=0;i<len;i++){
		cout<<ht[i]<<endl;
	}
	hipFree(dx);
	hipFree(dt);
	hipFree(d_roots);
	hipHostFree(hx);
	hipHostFree(ht);
	hipHostFree(h_roots);
	return 0;
}
