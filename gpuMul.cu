/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
int main(){
	int len=64;
	uint32 *hx,*dx;
	uint64 *ht,*dt;

	hipHostMalloc(&hx,len*sizeof(uint32));
	hipHostMalloc(&ht,len*sizeof(uint64));

	hipMalloc(&dx,len*sizeof(uint32));
	hipMalloc(&dt,len*sizeof(uint64));

  for(int i=0;i<4;i++){
		hx[i]=11;		//32 bits random number
	 	ht[i]=0;
		//cout<<"test"<<hx[i];   //for test the function "rand"
	}
  for(int i=4;i<len;i++){
	  	hx[i]=22;
	  	ht[i]=0;
  }
  	hipMemcpy(dx,hx,len*sizeof(uint32),hipMemcpyHostToDevice);
  	ntt_64_1(dt,dx);
	hipMemcpy(ht,dt,len*sizeof(uint64),hipMemcpyDeviceToHost);

	for(int i=0;i<len;i++){
		cout<<ht[i]<<endl;
	}
	hipFree(dx);
	hipFree(dt);
	hipHostFree(hx);
	hipHostFree(ht);
	return 0;
}
