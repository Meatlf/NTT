/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
#define len 256
#define len_64K 65536
int main(){
	float time;
	const ZZ P=to_ZZ(0xffffffff00000001);
	const ZZ root_256=to_ZZ((uint64)14041890976876060974);
	const ZZ root_64K=to_ZZ((uint64)15893793146607301539);
	uint32 *hx,*dx;
	uint64 *ht,*dt;
	uint64 *hy,*dy;

	uint64 *h_roots,*d_roots;
  uint64 *h_roots_64K,*d_roots_64K;
  
	hipHostMalloc(&hx,len_64K*sizeof(uint32));
	hipHostMalloc(&ht,len_64K*sizeof(uint64));
	hipHostMalloc(&hy,len_64K*sizeof(uint64));

	hipHostMalloc(&h_roots,len*sizeof(uint64));
  hipHostMalloc(&h_roots_64K,len_64K*sizeof(uint64));

	hipMalloc(&dx,len_64K*sizeof(uint32));
	hipMalloc(&dt,len_64K*sizeof(uint64));
	hipMalloc(&dy,len_64K*sizeof(uint64));

	hipMalloc(&d_roots,len*sizeof(uint64));
	hipMalloc(&d_roots_64K,len_64K*sizeof(uint64));
  for(int i=0;i<len_64K;i++){
		hx[i]=i;	
	 	ht[i]=0;
		hy[i]=0;
	}
  for(int i=0;i<16;i++){
		for(int k=0;k<16;k++){
				conv(h_roots[16*i+k],PowerMod(root_256,i*k,P));
		//test:		cout<<h_roots[16*i+k]<<endl;
		}
	}
	for(int i=0;i<256;i++){
		for(int k=0;k<256;k++){
				conv(h_roots_64K[256*i+k],PowerMod(root_64K,i*k,P));
		}
	}
  	hipMemcpy(dx,hx,len_64K*sizeof(uint32),hipMemcpyHostToDevice);
  	hipMemcpy(d_roots,h_roots,len*sizeof(uint64),hipMemcpyHostToDevice);
		hipMemcpy(d_roots_64K,h_roots_64K,len_64K*sizeof(uint64),hipMemcpyHostToDevice);
  	time=ntt_64K(dy,dt,dx,d_roots,d_roots_64K);
  	hipMemcpy(hy,dy,len_64K*sizeof(uint64),hipMemcpyDeviceToHost);
	
	cout<<time<<endl;

	for(int i=0;i<len_64K;i++){
		cout<<hy[i]<<endl;
	}
	hipFree(dx);
	hipFree(dt);
	hipFree(dy);
	hipFree(d_roots);
	hipFree(d_roots_64K);
	hipHostFree(hx);
	hipHostFree(ht);
	hipHostFree(hy);
	hipHostFree(h_roots);
	hipHostFree(h_roots_64K);
	return 0;
}
