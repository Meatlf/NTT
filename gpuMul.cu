/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : ttz 
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

		
int main(){
	int len=16;
	uint64 *hx,*dx;

	hipHostMalloc(&hx,len*sizeof(uint64));

	hipMalloc(&dx,len*sizeof(uint64));

  for(int i=0;i<4;i++){
		hx[i]=11;		
	}
	for(int i=4;i<16;i++){
		hx[i]=33;
	}
  	hipMemcpy(dx,hx,len*sizeof(uint64),hipMemcpyHostToDevice);
	ntt_16_1(dx);
	hipMemcpy(hx,dx,len*sizeof(uint64),hipMemcpyDeviceToHost);

	for(int i=0;i<len;i++){
		cout<<hx[i]<<endl;
	}
	hipFree(dx);
	hipHostFree(hx);
	return 0;
}
