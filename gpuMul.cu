/*
 ============================================================================
 Name        : gpuMul.cu
 Author      : jiyang
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include "NTT.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <stdint.h>
#include <stdlib.h>
#include "ModP.h"
#include "kernel.h"
#include <NTL/ZZ.h>
#include <NTL/ZZX.h>

using namespace NTL;

void outputGPU(uint64_t* input, uint64_t length){
	uint64_t* temp = (uint64_t*)malloc(sizeof(uint64_t) * length);
	hipMemcpy(input, temp, length * sizeof(uint64_t), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < length; i++){
		cout << temp[i] << ", ";
		if(i == length - 1){
			cout << endl;
		}
	}
	free(temp);
	return;
}


void outputGPU(uint32_t* input, uint64_t length){
	uint32_t* temp = (uint32_t*)malloc(sizeof(uint32_t) * length);
	hipMemcpy(input, temp, length * sizeof(uint32_t), hipMemcpyDeviceToHost);

	for(int i = 0; i < length; i++){
		cout << temp[i] << ", ";
		if(i == length - 1){
			cout << endl;
		}
	}
	free(temp);
	return;
}

int main(void)
{

	int length = 65536;
	int size = length / 2;

	uint32_t* x = (uint32_t*)malloc(sizeof(uint32_t) * length);
	uint32_t* x_gpu;
	hipMalloc((void**)&x_gpu, length * sizeof(uint32_t));

	uint32_t* y = (uint32_t*)malloc(sizeof(uint32_t) * length);
	uint32_t* y_gpu;
	hipMalloc((void**)&y_gpu, length * sizeof(uint32_t));

	uint64_t* X = (uint64_t*)malloc(sizeof(uint32_t) * length);
	uint64_t* X_gpu;
	hipMalloc((void**)&X_gpu, length * sizeof(uint64_t));


	uint64_t* Y = (uint64_t*)malloc(sizeof(uint64_t) * length);
	uint64_t* Y_gpu;
	hipMalloc((void**)&Y_gpu, length * sizeof(uint64_t));

	uint32_t* z = (uint32_t*)malloc(sizeof(uint32_t) * length);
	uint64_t* Z_gpu;
	hipMalloc((void**)&Z_gpu, length * sizeof(uint64_t));
	uint32_t* z_gpu;
	hipMalloc((void**)&z_gpu, length * sizeof(uint32_t));

	for(int i = 0; i < size; i++){
		x[i] = 16777215;
		y[i] = 16777215;
	}

	hipMemcpy(x, x_gpu, length * sizeof(uint32_t), hipMemcpyHostToDevice);
	hipMemcpy(y, y_gpu, length * sizeof(uint32_t), hipMemcpyHostToDevice);

	initNtt(length);

	_ntt(X_gpu, x_gpu, 0, 0, length);

	_ntt(Y_gpu, y_gpu, 0, 0, length);

	dotMul(X_gpu, Y_gpu, Z_gpu, length);

	free(x);free(y);free(X);free(Y);free(z);
	hipFree(x_gpu);hipFree(y_gpu);hipFree(X_gpu);hipFree(Y_gpu);hipFree(Z_gpu);hipFree(z_gpu);

	return 0;
}
